#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/zmath.cuh>

namespace at { namespace native {

// We manually overload abs because std::abs does not work with thrust::complex types and ROCm.
template<typename scalar_t>
__host__ __device__ static inline scalar_t abs_wrapper(scalar_t v) {
  return ::abs(v);
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> abs_wrapper(thrust::complex<T> v) {
  return thrust::abs(v);
}

__host__ __device__ static inline uint8_t abs_wrapper(uint8_t v) {
  return v;
}

__host__ __device__ static inline bool abs_wrapper(bool v) {
  return v;
}

void abs_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Bool, iter.dtype(), "abs_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "abs_cuda", [&] {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
        return abs_wrapper(a);
      });
    });
  });
}

REGISTER_DISPATCH(abs_stub, &abs_kernel_cuda);

}} // namespace at::native
